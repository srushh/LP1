
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;

__global__ void addition(int *a, int*b, int n)
{
  int tid=threadIdx.x;
  int sum=0;
  for(int i=0;i<n;i++)
  {
      sum+=a[i];
  } 
  b[tid]=sum; 
}

int main()
{
  int n=10;
  int *a=(int*)malloc(n*sizeof(int));
  hipEvent_t start, end;
  for(int i=0;i<n;i++)
  {
      a[i]=i+1;
  }
  hipEventCreate(&start);
  hipEventCreate(&end);
  int *dev_a,  *dev_b;
  int size=n*sizeof(int);
  hipMalloc(&dev_a,size);
  hipMalloc(&dev_b,sizeof(int));
  hipMemcpy(dev_a,a,size,hipMemcpyHostToDevice);
  hipEventRecord(start);
  addition<<<1, n>>>(dev_a, dev_b, n);
  int *sum=(int *)malloc(sizeof(int));
  hipEventRecord(end);
   hipEventSynchronize(end);
  float time=0;
  hipEventElapsedTime(&time, start, end);
  hipMemcpy(sum, dev_b, sizeof(int),hipMemcpyDeviceToHost);
  cout<<"\nAddition is : "<<sum[0];
  float avg=0;
  avg=sum[0]/(n*1.0);
  cout<<"\nMean is : "<<avg;
  cout<<"\nTime taken : "<<time;
  return 0;
}
