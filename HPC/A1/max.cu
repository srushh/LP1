
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;

__global__ void maximum(int *a, int*b, int n)
{
  int tid=threadIdx.x;
  int max=-9999;
  for(int i=0;i<n;i++)
  {
    if(max<a[i])
      max=a[i];
  } 
  b[tid]=max; 
}

int main()
{
  int n=1000;
  int *a=(int*)malloc(n*sizeof(int));
  hipEvent_t start, end;
  for(int i=0;i<n;i++)
  {
      a[i]=i+1;
  }
  hipEventCreate(&start);
  hipEventCreate(&end);
  int *dev_a,  *dev_b;
  int size=n*sizeof(int);
  hipMalloc(&dev_a,size);
  hipMalloc(&dev_b,sizeof(int));
  hipMemcpy(dev_a,a,size,hipMemcpyHostToDevice);
  hipEventRecord(start);
  maximum<<<1, n>>>(dev_a, dev_b, n);
  int *ans=(int *)malloc(sizeof(int));
  hipEventRecord(end);
  float time=0;
  hipEventSynchronize(end);
  hipEventElapsedTime(&time, start, end);
  hipMemcpy(ans, dev_b, sizeof(int),hipMemcpyDeviceToHost);
  cout<<"\nMaximum value is : "<<ans[0];
  cout<<"\nTime taken : "<<time;
  return 0;
}
