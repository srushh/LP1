
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;

__global__ void average(int *a, float *b, int n)
{
  int tid=threadIdx.x;
  int sum=0;
  for(int i=0;i<n;i++)
  {
      sum+=a[i];
  } 
 float mean=sum/(n*1.0);
  b[tid]=mean; 
}

__global__ void standardDev(int *a, float *b, float mean, int n)
{
    int tid=blockIdx.x;
    b[0]=0.0;
    for(int i=0;i<n;i++)
    {
        b[0] += (a[i] - mean) * (a[i] - mean);
    }
  b[0]=b[0]/n;
}

int main()
{
  int n=10;
  int *a=(int*)malloc(n*sizeof(int));
  hipEvent_t start, end;
  for(int i=0;i<n;i++)
  {
      a[i]=i+1;
  }
  hipEventCreate(&start);
  hipEventCreate(&end);
  int *dev_a;
  float  *dev_b;
  int size=n*sizeof(int);
  hipMalloc(&dev_a,size);
  hipMalloc(&dev_b,sizeof(float));
  hipMemcpy(dev_a,a,size,hipMemcpyHostToDevice);
  hipEventRecord(start);
  average<<<1, n>>>(dev_a, dev_b, n);
  float *mean=(float *)malloc(sizeof(float));
  hipEventRecord(end);
  hipEventSynchronize(end);
  float time=0;
  hipEventElapsedTime(&time, start, end);
  hipMemcpy(mean, dev_b, sizeof(float),hipMemcpyDeviceToHost);
  cout<<"\nMean is : "<<mean[0];
  float *std=(float*)malloc(sizeof(float));
  standardDev<<<n,1>>>(dev_a, dev_b, mean[0], n);
  hipMemcpy(std, dev_b, sizeof(float), hipMemcpyDeviceToHost);
  cout<<"\nStandard Deviation is : "<<sqrt(std[0])<<endl;
  cout<<"\nTime taken : "<<time;
  return 0;
}
